#include "hip/hip_runtime.h"
#include <stdio.h>

__device__ const char *STR = "HELLO WORLD";
const char STR_LENGTH = 12;

__global__ void hello(){
  printf("%c\n", STR[threadIdx.x % STR_LENGTH]);
}

int main(void){
  int num_threads = STR_LENGTH;
  int num_blocks = 1;
  hello<<<num_blocks, num_threads>>>()
  //  等待所有的host 线程中执行的命令执行完毕
  hipDeviceSynchronize();
  return 0;
}
