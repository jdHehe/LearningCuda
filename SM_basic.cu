#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, size_t size);

__global__ void addKernel(int *c, const int *a){
    int i = threadIdx.x;
    extern __shared__ int seme [];
    seme[i] = a[i];
    __syncthreads();  //同一个块的线程同步
    if(i==0){ //第一个线程进行二次方
        c[0] = 0;
        for (int d=0; d<5; d++){
            c[0] += seme[d] * seme [d];
        }
    }
    if(i==1){
        c[1] = 0;
        for (int d=0; d<5; d++){
            c[1] += seme[d];
        }
    }
    if(i==2){
        c[2] = 1;
        for(int d=0; d<5; d++){
            c[2] *= seme[d];
        }
    }
}

int main(){
    const int arraySize = 5;
    const int a[arraySize] = {1, 2, 3, 4, 5};
    int c[arraySize] = {0};
    hipError_t cudaStatus = addWithCuda(c, a, arraySize);
    if (cudaStatus != hipSuccess){
        fprintf(stderr, "addWithCuda 失败");
        return 1;
    }
    printf("\t1+2+3+4+5 = %d\n\t1^2+2^2+3^2+4^2+5^2 = %d\n\t1*2*3*4*5 = %d\n\n\n\n\n\n", c[1], c[0], c[2]);
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess){
        fprintf(stderr, "hipDeviceReset 失败");
        return 1;
    }
    return 0;
}

hipError_t addWithCuda(int *c,const int *a, size_t size){
    int *dev_a = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;
    
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess){
        fprintf(stderr, "cuda 分配内存失败");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess){
        fprintf(stderr, "cuda 分配内存失败");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess){
        fprintf(stderr, "从Device向Hostcopy数据失败");
        goto Error;
    }

    addKernel<<<1, size, size * sizeof(int), 0>>>(dev_c, dev_a);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess){
        fprintf(stderr, "cuda线程同步异常");
        goto Error;
    }

    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess){
        fprintf(stderr, "从Device向Hostcopy数据失败");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    return cudaStatus;
}