#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void vector_add(int *a, int *b, int *c){
  int index =  blockIdx.x * blockDim.x + threadIdx.x;
  c[index] = a[index] + b[index] ;
}

#define N(2048*2048)
#define THREADS_PER_BLOCK 512

int main(){
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;

  int size = N * sizeof( int );
  //
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_c, size);

  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(size);

  for (int i=0; i<N; i++){
    a[i] = b[i] = i;
    c[i] = 0;
  }

  // 将数据从host内存 拷贝到 gup的memory中
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  add<<, THREADS_PER_BLOCK>>(d_a, d_b, d_c);
  // 将GPU计算结束的数据，拷贝到主机的内存
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  printf("c[0]= %d\n", 0, c[0])
  printf("c[%d] = %d", N-1, c[N-1]);

  // 释放内存
  free(a);
  free(b);
  free(c);
  hipFree( d_a );
  hipFree( d_b );
  hipFree( d_c );

  return 0;
}
